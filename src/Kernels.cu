#include "hip/hip_runtime.h"
#include <vector>
#include <hip/hip_runtime.h>
#include <random>
#include <set>

__global__ void partition_mesh_kernel(int* visitedNodes, int dim_d, int* neighbors, int dim_ne, int* indices, int* frontier,
                                      int dim_f, int* new_frontier, int* dim_nf){
    if (threadIdx.x + blockDim.x * blockIdx.x < dim_f) {
        int vertex = frontier[threadIdx.x + blockDim.x * blockIdx.x];
        int end = vertex == dim_d - 1 ? dim_ne : indices[vertex + 1];
        for (int i = indices[vertex]; i < end; i++) {
            int v = neighbors[i];
            int old = atomicCAS(&visitedNodes[v], 0, visitedNodes[vertex]);
            if (old == 0) {
                int index = atomicAdd(dim_nf, 1);
                new_frontier[index] = v;
            }
        }
    }
}

void partition_mesh_host(std::vector<int>* neighbors, std::vector<int>* indices, int n_sub){
    int *visitedNodes_cpu, *visitedNodes_gpu, *frontier_cpu, *frontier_gpu,
                *new_frontier_cpu, *new_frontier_gpu, *neighbors_gpu, *indices_gpu;
    int dim_d, dim_ne, dim_f, *dim_nf;
    dim_d = indices->size();
    dim_ne = neighbors->size();
    dim_f = n_sub;
    std::random_device dev;
    std::mt19937 rng(dev());
    std::uniform_int_distribution<std::mt19937::result_type> dist(0,dim_d - 1);
    std::vector<std::set<int>> v(n_sub);
    for(int i = 0; i < n_sub; i++) {
        new_frontier_cpu[i] = (int)dist(rng);
    }
    for(int i = 0; i < dim_d; i++){
        visitedNodes_cpu[i] = 0;
    }

    //allocate and initialize visitedNodes
    hipMalloc(&visitedNodes_gpu, indices->size() * sizeof(int));
    hipMemcpy(visitedNodes_gpu, visitedNodes_cpu, dim_d * sizeof(int), hipMemcpyHostToDevice);
    //allocate frontier
    hipMalloc(&frontier_gpu, dim_d * sizeof(int));
    //allocate and initialize indices
    hipMalloc(&indices_gpu, dim_d * sizeof(int));
    hipMemcpy(indices_gpu, indices, dim_d * sizeof(int), hipMemcpyHostToDevice);
    //allocate and initialize neighbors
    hipMalloc(&neighbors_gpu, dim_ne * sizeof(int));
    hipMemcpy(neighbors_gpu, neighbors, dim_ne * sizeof(int), hipMemcpyHostToDevice);
    //allocate and initialize new_frontier (which will be assigned to frontier in the while loop)
    hipMalloc(&new_frontier_gpu, dim_d * sizeof(int));
    hipMemcpy(new_frontier_gpu, new_frontier_cpu, dim_d * sizeof(int), hipMemcpyHostToDevice);
    //allocate and initialize dim_new_frontier (which will be assigned to dim_frontier in the while loop)
    hipMalloc(&dim_nf, sizeof(int));
    hipMemset(&dim_nf, n_sub, sizeof(int));

    while(dim_f != 0){
        //copy new_frontier into frontier
        hipMemcpy(frontier_gpu, new_frontier_gpu, dim_d * sizeof(int), hipMemcpyDeviceToDevice);
        //copy dim_nf into dim_f
        hipMemcpy(&dim_f, dim_nf, sizeof(int), hipMemcpyDeviceToHost);
        //set dim_nf to zero
        hipMemset(&dim_nf, 0, sizeof(int));
        partition_mesh_kernel<<<5,4>>>(visitedNodes_gpu, dim_d, neighbors->data(), dim_ne, indices, frontier_gpu,
                dim_f, new_frontier_gpu, dim_nf);
    }

}