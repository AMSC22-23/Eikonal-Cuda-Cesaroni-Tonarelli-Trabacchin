#include "hip/hip_runtime.h"
#ifndef KERNELS
#define KERNELS
#include <vector>
#include <hip/hip_runtime.h>
#include <random>
#include <set>
#include <array>
//#include "../localProblem_alt2/include/Phi.hpp"
//#include "../localProblem_alt2/include/solveEikonalLocalProblem.hpp"
#include "Mesh.cuh"
#include "LocalSolver.cuh"
#include "CudaEikonalTraits.cuh"
constexpr int D = 3;
/*
using VectorExt = typename Eikonal::Eikonal_traits<3, 2>::VectorExt;
using Matrix = typename Eikonal::Eikonal_traits<D,2>::AnisotropyM;
using VectorV = typename Eigen::Matrix<double,4,1>;
*/




template <typename Float>
__global__ void setSolutionsToInfinity(Float* solutions_dev, Float infinity_value, size_t size_sol){
    unsigned int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    if(threadId < size_sol){
        solutions_dev[threadId] = infinity_value;
    }
}

template <typename Float>
__global__ void setSolutionsSourcesAndDomains(Float* solutions_dev, int* source_nodes_dev, int* active_domains_dev, int* partitions_vertices_dev, int partitions_number, size_t size_sources){
    unsigned int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    if(threadId < size_sources) {

        solutions_dev[source_nodes_dev[threadId]] = 0.0;
        bool found = false;
        for(int i = 0; i < partitions_number && !found; i++) {
            if (source_nodes_dev[threadId] <= partitions_vertices_dev[i]) {
                active_domains_dev[i] = 1; //there is no need to use atomic operations since if multiple threads try to access the memory location they all write 1
                found = true;
            }
        }
    }
}

template <typename Float>
__global__ void domainSweep(int domain_id, const int*  __restrict__ partitions_vertices_dev, int* __restrict__ partitions_tetra_dev, Float* __restrict__ geo_dev, const int* __restrict__ tetra_dev,
                            const TetraConfig* __restrict__ shapes_dev, const int* __restrict__ ngh, const Float* __restrict__ M_dev, Float* __restrict__ solutions_dev, int* __restrict__ active_domains_dev,
                            int num_partitions, int num_vertices, int num_tetra, int shapes_size, Float infinity_value, Float tol){
                                
    using VectorExt = typename CudaEikonalTraits<Float, D>::VectorExt;
    using VectorV = typename CudaEikonalTraits<Float, D>::VectorV;
    using Matrix = typename CudaEikonalTraits<Float, D>::Matrix;
                                
    unsigned int nodeIdDomain = threadIdx.x + blockIdx.x * blockDim.x;
    /*if(nodeIdDomain == 0) {
        printf("searching domain %d\n", domain_id);
    }*/
    unsigned int nodeIdGlobal = ((domain_id != 0) ? (partitions_vertices_dev[domain_id - 1]) : 0) + nodeIdDomain;
    
    //std::array<VectorExt, 4> coordinates;
    VectorExt coordinates[4];
    VectorV values;
    const Float* M;
    Float minimum_sol = infinity_value;
    // each thread takes a node and compute the solution looping over all its associated tetrahedra
    if (nodeIdGlobal < num_vertices){
        for(int i = ngh[nodeIdGlobal]; i < ((nodeIdGlobal != num_vertices - 1) ? ngh[nodeIdGlobal+1]: shapes_size); i++){
            // call local solver on tetra[shapes_dev[i].tetra_index] using configuration shapes_dev[i].tetra_config
            for(int j = 0; j < D + 1; j++){
                for(int k = 0; k < D; k++) {
                    coordinates[j][k] =  geo_dev[D * tetra_dev[(D+1) * shapes_dev[i].tetra_index + j] + k];
                }
                values[j] = solutions_dev[tetra_dev[(D+1) * shapes_dev[i].tetra_index + j]];
            }
            M = M_dev + shapes_dev[i].tetra_index * 6;
            auto [sol, lambda1, lambda2] = LocalSolver<D, Float>::solve(coordinates, values, M, D+1-shapes_dev[i].tetra_config);
            //printf("sol = %f\n", sol);
            if(sol < minimum_sol) {
                minimum_sol = sol;
            }
        }

        if(-minimum_sol + solutions_dev[nodeIdGlobal] > tol) {
            //printf("min sol: %f %f\n",minimum_sol,  solutions_dev[nodeIdGlobal]);
            atomicExch(&solutions_dev[nodeIdGlobal], minimum_sol);
            //solutions_dev[nodeIdGlobal] = minimum_sol;
            
            
            bool found = false;
            for (int i = ngh[nodeIdGlobal]; i < ((nodeIdGlobal != num_vertices - 1) ? ngh[nodeIdGlobal+1]: shapes_size); i++) { //for each tetra associated with the node
                for (int j = 0; j < D + 1; j++) { //for each vertex in the tetra
                    found = false;
                    for (int k = 0; k < num_partitions && !found; k++) { // activate the domains associated with the vertices
                        if (tetra_dev[shapes_dev[i].tetra_index * (D + 1) + j] <= partitions_vertices_dev[k]) {
                            int old_active = atomicExch(&active_domains_dev[k], 1);
                            //active_domains_dev[k] = 1;
                            found = true;
                        }
                    }
                }
            }
        }
    }

}

#endif